
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>

#define PTX_MODULE_FILE "arith_latency_kernel.ptx"
#define BLOCK_SIZE 1024
#define BLOCK_NUMBER 30

static __device__ __inline__ uint32_t __mysmid()
{
    uint32_t smid;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(smid));
    return smid;
}

__global__
void WorkGroupCountTest(int *result, unsigned int *time)
{
    int iteration, a, b;
    int blockID;
    unsigned int *curTime;
    unsigned int start, end;
    
    blockID = blockIdx.y * blockDim.x + blockIdx.x;
    iteration = result[blockID];
    a = result[blockID];
    b = result[0];

    start = clock();
    for (int i = 0 ; i < iteration ; i ++)
    {
        a = a + b;
        b = b + a;
    }

    result[blockID] = b;
    end = clock();
   
    curTime = time + 5 * blockID;
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        curTime[0] = __mysmid();
        curTime[1] = blockID;
        curTime[2] = start;
        curTime[3] = end;
        curTime[4] = iteration;
    }
}

int main(int argc, char *argv[])
{
    // smid, start, end, blockid, iteration
    unsigned int *host_time = new unsigned int[5 * BLOCK_NUMBER];
    unsigned int *device_time;
    // loop iteration
    int *host_result = new int[BLOCK_NUMBER];
    int *device_result;
    hipSetDevice(1);

    srand(time(NULL));
    hipMalloc( (void **)(&device_time), 5 * BLOCK_NUMBER * sizeof(unsigned));
    hipMalloc( (void **)(&device_result), BLOCK_NUMBER * sizeof(int));

    for (int i = 0 ; i < BLOCK_NUMBER ; i ++)
    {
        host_result[i] = (rand() % 1000000 + 1);
        printf("%d\n", host_result[i]);
    }
    printf("\n");

    dim3 block(BLOCK_SIZE);
    dim3 grid(BLOCK_NUMBER);

    {
        hipMemcpy(device_result, host_result, BLOCK_NUMBER * sizeof(int), hipMemcpyHostToDevice);
        
        WorkGroupCountTest<<<grid, block>>>(device_result, device_time);
        
        hipMemcpy(host_time, device_time, 5 * BLOCK_NUMBER * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
        for (int i = 0 ; i < BLOCK_NUMBER ; i ++)
        {
            printf("smid = %3u, blockID = %3u, start = %-10u, end = %-10u, iteration = %-8u, duration = %-10u\n",
                    host_time[i * 5], host_time[i * 5 + 1], host_time[i * 5 + 2], host_time[i * 5 + 3], host_time[i * 5 + 4], host_time[i * 5 + 3] - host_time[i * 5 + 2]);
        }
    }
    hipFree(device_time);
    hipFree(device_result);

    delete [] host_time;
    delete [] host_result;
    return 0;
}
