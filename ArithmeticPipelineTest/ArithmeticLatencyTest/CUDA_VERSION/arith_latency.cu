#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cudaTestHelper.h"
#include <sys/time.h>

int main(int argc, char *argv[])
{
    hipError_t result;
    hipError_t error;
    CUDATestHelper helper;
    InitialAndParseArgs(argc, argv, helper);
    int kernelIter = 2;
    double executionTime;

    printf("Run testcase for %d iterations\n", helper.testIteration);
    for (unsigned int idx = 0 ; idx < helper.ptxKernelName.size() ; idx ++)
    {
    	executionTime = 0;
    	for (int iter = 0 ; iter < helper.testIteration ; iter ++)
    	{
	        hipModule_t ptxModule = 0;
	        hipFunction_t ptxFunction = 0;

	        error = hipMalloc( (void **)(&helper.deviceTime), 2 * sizeof(unsigned));
	        CHECK_CUDA_ERROR(error);
	        error = hipMalloc( (void **)(&helper.deviceData), helper.sizeOfData);
	        CHECK_CUDA_ERROR(error);

	        void *functionArgs[3] = {&helper.deviceData, &helper.deviceTime, &kernelIter};

	        result = hipModuleLoad(&ptxModule, helper.ptxFileName);
	        CHECK_CU_RESULT(result);

	        result = hipModuleGetFunction(&ptxFunction, ptxModule, helper.ptxKernelName[idx]);
	        CHECK_CU_RESULT(result);

	        error = hipMemcpy(helper.deviceData, helper.hostData, helper.sizeOfData, hipMemcpyHostToDevice);
	        CHECK_CUDA_ERROR(error);

	        result = hipModuleLaunchKernel(ptxFunction, GRID_X, GRID_Y, GRID_Z, BLOCK_X, BLOCK_Y, BLOCK_Z, 0, 0, functionArgs, 0);
	        CHECK_CU_RESULT(result);

	        error = hipMemcpy(helper.hostTime, helper.deviceTime, 2 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	        CHECK_CUDA_ERROR(error);

	        error = hipFree(helper.deviceTime);
	        CHECK_CUDA_ERROR(error);
	        error = hipFree(helper.deviceData);
	        CHECK_CUDA_ERROR(error);
		
	        executionTime += (helper.hostTime[1] - helper.hostTime[0]);
		}
		executionTime /= helper.testIteration;
        printf("%-50s %lf\n", helper.ptxKernelName[idx], executionTime);
    }

    return 0;
}
