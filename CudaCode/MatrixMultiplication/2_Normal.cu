#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>

#include "papi_wrapper.hpp"

#define TILE_W 32
#define Width 512
#define Element Width*Width
using namespace std;

__global__ 
void MatrixMulKernel(int* Md, int* Nd, int* Pd)
{
	//Block Index
	int by = blockIdx.y;
	int bx = blockIdx.x;
	//Thread Index
	int ty = threadIdx.y; //Row
	int tx = threadIdx.x; //Col
	
	int Row = by*TILE_W + ty;
	int Col = bx*TILE_W + tx;
	
	if((Row<Width) && (Col<Width))
	{
		//Pvalue is used to store the element of the matrix
		//That is computed by the thread
		int Pvalue = 0;

		for(int k=0 ; k<Width ; ++k)
			Pvalue += Md[Row*Width+k]*Nd[k*Width+Col];
        Pd[Row*Width + Col] = Pvalue;
	}

}

int main()
{
	cout << "----------------------------------------------Start" << endl;
	cout << "This is Normal Matrix Multiplication version" << endl;
	cout << "---------------------------------------------------" << endl;
	cout << "Grid Dimension : " << Width/TILE_W << "x" << Width/TILE_W << endl;
	cout << "Block Dimension : " << TILE_W << "x" << TILE_W << endl;
    cout << "Dimension : " << Width << "x" << Width << endl;
	cout << "Total Elements	: " << Element << endl;
	cout << "---------------------------------------------------" << endl;

    hipSetDevice(1);

	//Variables for Time
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	PAPIWrapper papi_ctrl;
    //papi_ctrl.AddEvent(2, strdup("cuda:::device:1:inst_executed"), strdup("cuda:::device:1:gld_inst_32bit"));
    papi_ctrl.AddEvent(2, strdup("cuda:::device:1:inst_executed"), strdup("cuda:::device:1:uncached_global_load_transaction"));

	int size = Element*sizeof(int);
	int* M = (int*)malloc(size);
	int* N = (int*)malloc(size);
	int* P = (int*)malloc(size);
	int* Temp_sum_array = (int*)malloc(size);
	int* Md;
	int* Nd;
	int* Pd;

	srand(time(0));
	for(int i=0 ; i<Element ; i++)
	{
		M[i] = rand()%100;
		N[i] = rand()%100;
		P[i] = 0;
	}
	
	hipEventRecord(start, 0);	
	//CPU Matrix Multiplication
	int Temp_sum = 0;
	for(int row=0 ; row<Width ; row++)
	{
		for(int col=0 ; col<Width ; col++)
		{	
			Temp_sum = 0;
			for(int n=0 ; n<Width ; n++)
			{
				Temp_sum += M[row*Width+n]*N[n*Width+col];
			}
			Temp_sum_array[row*Width+col] = Temp_sum;
		}
	}
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	float CPU_time;
	hipEventElapsedTime(&CPU_time, start, end);
	cout << "Matrix Multiplication by CPU : " << CPU_time/1000 << 's' << endl;
	//Finish

	/////////////////////////////////////////////////
	////////						 CUDA							//////////
	/////////////////////////////////////////////////
	hipEventRecord(start, 0);

	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd, size);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	float Memory_time;
	hipEventElapsedTime(&Memory_time, start, end);
	cout << "Time of Processing Memory		: " << Memory_time/1000 << 's' << endl;

	hipEventRecord(start, 0);

    papi_ctrl.Start();

	dim3 dimGrid(Width/TILE_W, Width/TILE_W);
	dim3 dimBlock(TILE_W, TILE_W);	
	MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	float GPU_time;
	hipEventElapsedTime(&GPU_time, start, end);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    papi_ctrl.Stop();
	
    cout << "Matrix Multiplication by GPU : " << GPU_time/1000 << 's' << endl;
	cout << "---------------------------------------------------" << endl;

	//Print CPU Result
	//cout << "CPU Result :" << endl;
	//for(int i=0 ; i<Element ; i++)
	//	cout << Temp_sum_array[i] << ", ";
	//cout << endl;
	
	//Print GPU Result
	//cout << "GPU Result :" << endl;
	//for(int i=0 ; i<Element ; i++)
	//	cout << P[i] << ", ";
	//cout << endl; 
	
	//Check Multiplication Result
	int check_flag = 0;
	for(int i=0 ; i<Element ; i++)
		if(Temp_sum_array[i] != P[i])
		{
			cout << "Wrong Point at : " << i << endl;
			cout << "CPU Results is : " << Temp_sum_array[i] << endl;
			cout << "GPU Results is : " << P[i] << endl;
			check_flag = 1;
			break;
		}	

	if(check_flag == 1)
		cout << "Wrong Result" << endl;
	else if(check_flag == 0)
		cout << "Correct Result" << endl;
	//Finish	

	//Compare CPU_time and GPU_time
	if(CPU_time > GPU_time)
	{
		cout << "GPU is faster" << endl;
		float SpeedUp = CPU_time/GPU_time;
		cout << "Speedup : " << SpeedUp << "x" << endl;
	}
	else
		cout << "CPU is faster" << endl;
	//Finish
	cout << "------------------------------------------------End" << endl;


    free(M);
    free(N);
    free(P);
    free(Temp_sum_array);
    /*
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipEventDestroy(start);
	hipEventDestroy(end);
	*/
    return EXIT_SUCCESS;
}
