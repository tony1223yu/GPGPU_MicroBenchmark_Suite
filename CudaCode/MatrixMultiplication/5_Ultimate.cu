
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#define TILE_W 16
#define Width 1600
#define Element Width*Width
using namespace std;

__global__
void MatrixMulKernel_SharedMemory(int* Md, int* Nd, int* Pd)
{	
	//Block Index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	//Thread Index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	__shared__ int Mds_1[TILE_W][TILE_W];
	__shared__ int Mds_2[TILE_W][TILE_W];
        __shared__ int Nds_1[TILE_W][TILE_W];
	__shared__ int Nds_2[TILE_W][TILE_W];

	int Row = by*TILE_W + ty;
	int Col = bx*TILE_W + tx;
	
	int Pvalue_1 = 0;
	int Pvalue_2 = 0;
	int Pvalue_3 = 0;
	int Pvalue_4 = 0;

	for(int m=0 ; m<Width/TILE_W ; ++m)
	{
		int Mds_1_Index = Row*Width + m*TILE_W + tx;
		int Mds_2_Index = (Row+Width/2)*Width + m*TILE_W + tx;
		int Nds_1_Index = (m*TILE_W + ty)*Width + Col;
		int Nds_2_Index = (m*TILE_W + ty)*Width + Col + Width/2;
		
		Mds_1[ty][tx] = Md[Mds_1_Index];
		Mds_2[ty][tx] = Md[Mds_2_Index];
		Nds_1[ty][tx] = Nd[Nds_1_Index];
		Nds_2[ty][tx] = Nd[Nds_2_Index];

		//Synchronize to make sure the sub-matirces are loaded
		//before starting the computation
		__syncthreads();
		
		//Each thread computes one element of the block sub-matrix
		for(int k=0 ; k<TILE_W ; ++k)
		{
			Pvalue_1 += Mds_1[ty][k] * Nds_1[k][tx];
			Pvalue_2 += Mds_1[ty][k] * Nds_2[k][tx];
			Pvalue_3 += Mds_2[ty][k] * Nds_1[k][tx];
			Pvalue_4 += Mds_2[ty][k] * Nds_2[k][tx];
		}
	
		//Synchronize to make sure that the preceding
		//computation is done before loading two new
		//sub-matrices of M and N in the next iteration
		__syncthreads();
    }

	Pd[Row*Width + Col] = Pvalue_1;
	Pd[Row*Width + Col + Width/2] = Pvalue_2;
	Pd[(Row+Width/2)*Width + Col] = Pvalue_3;
	Pd[(Row+Width/2)*Width + Col + Width/2] = Pvalue_4;
}

int main()
{	
	cout << "----------------------------------------------Start" << endl;	
	cout << "This is Ultimate version" << endl;
	cout << "---------------------------------------------------" << endl;
	cout << "Grid  Dimension : " << (Width/TILE_W)/2 << "x" << (Width/TILE_W)/2 << endl;
	cout << "Block Dimension : " << TILE_W << "x" << TILE_W << endl;
	cout << "Dimension       : " << Width << "x" << Width << endl;
        cout << "Total Elements  : " << Element << endl;
	cout << "---------------------------------------------------" << endl;	

	//Variables for Time
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	int size = Element*sizeof(int);
	int* M = (int*)malloc(size);
	int* N = (int*)malloc(size);
	int* P = (int*)malloc(size);
	int* Temp_sum_array = (int*)malloc(size);
	int* Md;
        int* Nd;
        int* Pd;

	srand(time(0));
	for(int i=0 ; i<Element ; i++)
	{
		M[i] = rand()%1000;
		N[i] = rand()%1000;
		P[i] = 0;
	}

	//Print M
	/*
        cout << "M Matrix : " << endl;
        for(int i=0 ; i<Width ; i++)
        {
                for(int j=0 ; j<Width ; j++)
                        cout << M[i] << ", ";
                cout << endl;
        }
	cout << endl;
	//Print N
        cout << "N Matrix : " << endl;
        for(int i=0 ; i<Width ; i++)
        {
                for(int j=0 ; j<Width ; j++)
                        cout << N[i] << ", ";
                cout << endl;
        }*/
	
	hipEventRecord(start, 0);
	//CPU Matirx Multiplication
	int Temp_sum = 0;
	for(int row=0 ; row<Width ; row++)
	{
		for(int col=0 ; col<Width ; col++)
		{
			Temp_sum = 0;
			for(int n=0 ; n<Width ; n++)
			{
				Temp_sum += M[row*Width+n]*N[n*Width+col];
			}
			Temp_sum_array[row*Width+col] = Temp_sum;
		}
	}
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	float CPU_time;
	hipEventElapsedTime(&CPU_time, start, end);
        cout << "Matrix Multiplication by CPU : " << CPU_time/1000 << 's' << endl;
	//Finish
	
	/////////////////////////////////////////////////
	////////             CUDA              //////////
	/////////////////////////////////////////////////
	hipEventRecord(start, 0);

	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd, size);
	
	hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float Memory_time;
        hipEventElapsedTime(&Memory_time, start, end);
	cout << "Time of Processing Memory    : " << Memory_time/1000 << 's' << endl;

	hipEventRecord(start, 0);

	dim3 dimGrid((Width/TILE_W)/2, (Width/TILE_W)/2);
	dim3 dimBlock(TILE_W, TILE_W);
	size_t SharedMemoryBytes = sizeof(int)*TILE_W*TILE_W*4;
	MatrixMulKernel_SharedMemory<<<dimGrid, dimBlock, SharedMemoryBytes>>>(Md, Nd, Pd);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
        float GPU_time;
        hipEventElapsedTime(&GPU_time, start, end);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
 
        cout << "Matrix Multiplication by GPU : " << GPU_time/1000 << 's' << endl;
	cout << "---------------------------------------------------" << endl;	

	//Print GPU Result
	/*cout << "GPU Result : (With Tiling Algorithm)" << endl; 
	for(int i=0 ; i<Width ; i++)
	{
		for(int j=0 ; j<Width ; j++)
			cout << P[i*Width + j] << ", ";
		cout << endl;
	}
	
	//Print CPU Result
        cout << "CPU Result : " << endl;
        for(int i=0 ; i<Width ; i++)
	{
		for(int j=0 ; j<Width ; j++)
			cout << Temp_sum_array[i*Width + j] << ", ";
		cout << endl;
	}*/

	//Check Multiplication Result
	int check_flag = 0;
        for(int i=0 ; i<Element ; i++)
                if(Temp_sum_array[i] != P[i])
                {
			cout << "Wrong Point at : " << i << endl;
			cout << "CPU Results is : " << Temp_sum_array[i] << endl;
			cout << "GPU Results is : " << P[i] << endl;
                        check_flag = 1;
                        break;
                }

        if(check_flag == 1)
                cout << "Wrong Result" << endl;
        else if(check_flag == 0)
                cout << "Correct Result" << endl;
	//Finish

	//Compare CPU_time and GPU_time
	if(CPU_time > GPU_time)
	{
		cout << "GPU is faster" << endl;
		float SpeedUp = CPU_time/GPU_time;
		cout << "Speedup : " << SpeedUp << "x" << endl;
	}
	else
		cout << "CPU is faster" << endl;	
	//Finish
	cout << "------------------------------------------------End" << endl;
	
	free(M);
	free(N);
	free(P);
	free(Temp_sum_array);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipEventDestroy(start);
	hipEventDestroy(end);
	
	return EXIT_SUCCESS;
}
