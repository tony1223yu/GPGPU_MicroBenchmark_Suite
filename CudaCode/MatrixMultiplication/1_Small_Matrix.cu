
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#define Width 32
#define Element 1024
using namespace std;

__global__ 
void MatrixMulKernel(int* Md, int* Nd, int* Pd)
{
	//Thread Index
	int ty = threadIdx.y; //Row
	int tx = threadIdx.x; //Col
	
	//Pvalue is used to store the element of the matrix
        //That is computed by the thread
	int Pvalue = 0;

	if((ty<Width) && (tx<Width))
	{
		for(int k=0 ; k<Width ; ++k)
			Pvalue += Md[ty*Width+k]*Nd[k*Width+tx];
    }

    Pd[ty*Width + tx] = Pvalue;
}

int main()
{
	cout << "----------------------------------------------Start" << endl;	
	cout << "This is CPU faster than GPU Version" << endl;
	cout << "---------------------------------------------------" << endl;
	cout << "Grid  Dimension : " << "1" << endl;
	cout <<	"Block Dimension : " << Width << endl;
	cout << "Dimension       : " << Width << endl;
	cout << "Total Elements  : " << Element << endl;
	cout << "---------------------------------------------------" << endl;
	//Variables for Time
	hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        int size = Element*sizeof(int);
        int* M = (int*)malloc(size);
        int* N = (int*)malloc(size);
        int* P = (int*)malloc(size);
        int* Temp_sum_array = (int*)malloc(size);
        int* Md;
        int* Nd;
        int* Pd;	

	srand(time(0));
	for(int i=0 ; i<Element ; i++)
	{
		M[i] = rand()%100;
		N[i] = rand()%100;
		P[i] = 0;
	}
	
	hipEventRecord(start, 0);
	//CPU Matrix Multiplication
	int Temp_sum = 0;
	for(int row=0 ; row<Width ; row++)
	{
		for(int col=0 ; col<Width ; col++)
		{	
			Temp_sum = 0;
			for(int n=0 ; n<Width ; n++)
			{
				Temp_sum += M[row*Width+n]*N[n*Width+col];
			}
			Temp_sum_array[row*Width+col] = Temp_sum;
		}
	}
	hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float CPU_time;
        hipEventElapsedTime(&CPU_time, start, end);
        cout << "Matrix Multiplication by CPU : " << CPU_time/1000 << 's' << endl;
	//Finish

	/////////////////////////////////////////////////
        ////////             CUDA              //////////
        /////////////////////////////////////////////////
	hipEventRecord(start, 0);	
	
	hipMalloc((void**)&Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd, size);
	
	hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float Memory_time;
        hipEventElapsedTime(&Memory_time, start, end);
        cout << "Time of Processing Memory    : " << Memory_time/1000 << 's' << endl;

	hipEventRecord(start, 0);

	dim3 dimGrid(1, 1);
	dim3 dimBlock(Width, Width);
	MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
	
        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float GPU_time;
        hipEventElapsedTime(&GPU_time, start, end);

        hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

        cout << "Matrix Multiplication by GPU : " << GPU_time/1000 << 's' << endl;
        cout << "---------------------------------------------------" << endl;

	//Print CPU Result
	//cout << "CPU Result :" << endl;
	//for(int i=0 ; i<Element ; i++)
	//	cout << Temp_sum_array[i] << ", ";
	//cout << endl;
	
	//Print GPU Result
	//cout << "GPU Result :" << endl;
	//for(int i=0 ; i<Element ; i++)
	//	cout << P[i] << ", ";
	//cout << endl; 
	
	//Check Multiplication Result
	int check_flag = 0;
        for(int i=0 ; i<Element ; i++)
                if(Temp_sum_array[i] != P[i])
                {
                        cout << "Wrong Point at : " << i << endl;
                        cout << "CPU Results is : " << Temp_sum_array[i] << endl;
                        cout << "GPU Results is : " << P[i] << endl;
                        check_flag = 1;
                        break;
                }

	if(check_flag == 1)
		cout << "Wrong Result" << endl;
	else if(check_flag == 0)
		cout << "Correct Result" << endl;
	//Finish	

	//Compare CPU_time and GPU_time
        if(CPU_time > GPU_time)
                cout << "GPU is faster" << endl;
        else
                cout << "CPU is faster" << endl;
        //Finish
	cout << "------------------------------------------------End" << endl;
	
	free(M);
	free(N);
	free(P);
	free(Temp_sum_array);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipEventDestroy(start);
        hipEventDestroy(end);

	return EXIT_SUCCESS;
}
