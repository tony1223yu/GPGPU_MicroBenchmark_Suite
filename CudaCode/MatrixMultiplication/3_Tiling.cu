
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#define TILE_W 32
#define Width 1024
#define Element Width*Width
using namespace std;

__global__
void MatrixMulKernel_SharedMemory(int* Md, int* Nd, int* Pd)
{	
	//Block Index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	//Thread Index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	__shared__ int Mds[TILE_W][TILE_W];
  __shared__ int Nds[TILE_W][TILE_W];

	int Row = by*TILE_W + ty;
	int Col = bx*TILE_W + tx;
	
	int Pvalue = 0;

	for(int m=0 ; m<Width/TILE_W ; ++m)
	{
		Mds[ty][tx] = Md[Row*Width + m*TILE_W + tx];
		Nds[ty][tx] = Nd[(m*TILE_W + ty)*Width + Col];

		//Synchronize to make sure the sub-matirces are loaded
		//before starting the computation
		__syncthreads();
		
		//Each thread computes one element of the block sub-matrix
		for(int k=0 ; k<TILE_W ; ++k)
			Pvalue += Mds[ty][k] * Nds[k][tx];
	
		//Synchronize to make sure that the preceding
		//computation is done before loading two new
		//sub-matrices of M and N in the next iteration
		__syncthreads();
  }	
	
  Pd[Row*Width + Col] = Pvalue;
}

int main()
{	
	cout << "----------------------------------------------Start" << endl;
        cout << "This is Tiling version" << endl;
        cout << "---------------------------------------------------" << endl;
        cout << "Grid  Dimension : " << Width/TILE_W << "x" << Width/TILE_W << endl;
        cout << "Block Dimension : " << TILE_W << "x" << TILE_W << endl;
        cout << "Dimension       : " << Width << "x" << Width <<endl;
        cout << "Total Elements  : " << Element << endl;
        cout << "---------------------------------------------------" << endl;
	
	//Variables for Time
	hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);

        int size = Element*sizeof(int);
        int* M = (int*)malloc(size);
        int* N = (int*)malloc(size);
        int* P = (int*)malloc(size);
        int* Temp_sum_array = (int*)malloc(size);
        int* Md;
        int* Nd;
        int* Pd;

	srand(time(0));
	for(int i=0 ; i<Element ; i++)
	{
		M[i] = rand()%1000;
		N[i] = rand()%1000;
		P[i] = 0;
	}
	
	hipEventRecord(start, 0);
	//CPU Matirx Multiplication
	int Temp_sum = 0;
	for(int row=0 ; row<Width ; row++)
	{
		for(int col=0 ; col<Width ; col++)
		{
			Temp_sum = 0;
			for(int n=0 ; n<Width ; n++)
			{
				Temp_sum += M[row*Width+n]*N[n*Width+col];
			}
			Temp_sum_array[row*Width+col] = Temp_sum;
		}
	}
	hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float CPU_time;
        hipEventElapsedTime(&CPU_time, start, end);
        cout << "Matrix Multiplication by CPU : " << CPU_time/1000 << 's' << endl;
	//Finish
	
	/////////////////////////////////////////////////
	////////             CUDA              //////////
	/////////////////////////////////////////////////
	hipEventRecord(start, 0);

        hipMalloc((void**)&Md, size);
        hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
        hipMalloc((void**)&Nd, size);
        hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
        hipMalloc((void**)&Pd, size);

        hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float Memory_time;
        hipEventElapsedTime(&Memory_time, start, end);
        cout << "Time of Processing Memory    : " << Memory_time/1000 << 's' << endl;

        hipEventRecord(start, 0);

	dim3 dimGrid(Width/TILE_W, Width/TILE_W);
	dim3 dimBlock(TILE_W, TILE_W);
	size_t SharedMemoryBytes = sizeof(int)*TILE_W*TILE_W*2;
	MatrixMulKernel_SharedMemory<<<dimGrid, dimBlock, SharedMemoryBytes>>>(Md, Nd, Pd);

	hipEventRecord(end, 0);
        hipEventSynchronize(end);
        float GPU_time;
        hipEventElapsedTime(&GPU_time, start, end);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

        cout << "Matrix Multiplication by GPU : " << GPU_time/1000 << 's' << endl;
	cout << "---------------------------------------------------" << endl;	

	//Print GPU Result
	//cout << "GPU Result : (With Tiling Algorithm)" << endl; 
	//for(int i=0 ; i<Element ; i++)
	//	cout << P[i] << ", ";
	//cout << endl;
	
	//Print CPU Result
        //cout << "CPU Result : " << endl;
        //for(int i=0 ; i<Element ; i++)
        //      cout << Temp_sum_array[i] << ", ";
        //cout << endl;

	//Check Multiplication Result
	int check_flag = 0;
        for(int i=0 ; i<Element ; i++)
                if(Temp_sum_array[i] != P[i])
                {
                        cout << "Wrong Point at : " << i << endl;
                        cout << "CPU Results is : " << Temp_sum_array[i] << endl;
                        cout << "GPU Results is : " << P[i] << endl;
                        check_flag = 1;
                        break;
                }
        if(check_flag == 1)
                cout << "Wrong Result" << endl;
        else if(check_flag == 0)
                cout << "Correct Result" << endl;
	//Finish

	//Compare CPU_time and GPU_time
	if(CPU_time > GPU_time)
        {
                cout << "GPU is faster" << endl;
                float SpeedUp = CPU_time/GPU_time;
                cout << "Speedup : " << SpeedUp << "x" << endl;
        }
	else
		cout << "CPU is faster" << endl;	
	//Finish
	cout << "------------------------------------------------End" << endl;
	
	free(M);
	free(N);
	free(P);
	free(Temp_sum_array);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);
	hipEventDestroy(start);
        hipEventDestroy(end);
	
	return EXIT_SUCCESS;
}
