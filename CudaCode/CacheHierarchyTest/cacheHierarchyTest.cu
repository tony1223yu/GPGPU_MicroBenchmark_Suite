#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <iostream>
#include "papi_wrapper.hpp"

using namespace std;

__global__ void Processing(long* dataArray, long iter, long offset, int interval)
{
    long* currArray = dataArray + blockIdx.x * offset + threadIdx.x * interval;
    while (iter -- > 0)
    {
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
        currArray = (long *)(*currArray);
    }
    dataArray[blockIdx.x * offset + threadIdx.x * interval] = (long)(currArray);
}
__global__ void GeneratePattern(long* dataArray, int size, int stride, int interval)
{
    int idx = 0;
    long* currArray = dataArray + blockIdx.x * stride * size + threadIdx.x * interval;
    for (int i = 0 ; i < size - 1 ; i ++)
    {
        currArray[idx] = (long)(&currArray[idx + stride]);
        idx = idx + stride;
    }
    currArray[idx] = (ulong)(currArray);
}


/* Control struct */
struct CUDA_Ctrl
{
    int platform_id;
    int device_id;
    long dataByte;
    long iteration;
    int size;
    int stride;
    int interval;
    long offset;
    int globalSize;
    int localSize;

    CUDA_Ctrl() : platform_id(0), device_id(0), size(1), stride(1), iteration(1), globalSize(1), localSize(1), offset(1), interval(1) {} 
    ~CUDA_Ctrl() {}

} g_cuda_ctrl;


void CommandParser(int argc, char *argv[])
{
    char* short_options = strdup("p:d:s:S:i:o:g:l:i:v:");
    struct option long_options[] =
    {
        {"platformID", required_argument, NULL, 'p'},
        {"deviceID", required_argument, NULL, 'd'},
        {"iteration", required_argument, NULL, 'i'},
        {"size", required_argument, NULL, 'S'},
        {"stride", required_argument, NULL, 's'},
        {"interval", required_argument, NULL, 'v'},
        {"globalSize", required_argument, NULL, 'g'},
        {"localSize", required_argument, NULL, 'l'},
        /* option end */
        {0, 0, 0, 0}
    };
    int cmd;
    int optionIdx;
    while(1)
    {
        cmd = getopt_long(argc, argv, short_options, long_options, &optionIdx);

        /* finish parsing */
        if (cmd == -1)
            break;

        switch (cmd)
        {
            case 'g':
                g_cuda_ctrl.globalSize = atoi(optarg);
                break;

            case 'l':
                g_cuda_ctrl.localSize = atoi(optarg);
                break;

            case 'v':
                g_cuda_ctrl.interval = atoi(optarg);
                break;

            case 'S':
                g_cuda_ctrl.size = atoi(optarg);
                break;

            case 's':
                g_cuda_ctrl.stride = atoi(optarg);
                break;

            case 'i':
                g_cuda_ctrl.iteration = atol(optarg);
                break;

            case 'p':
                g_cuda_ctrl.platform_id = atoi(optarg);
                break;

            case 'd':
                g_cuda_ctrl.device_id = atoi(optarg);
                break;

            case '?':
                fprintf(stderr, "Unknown option -%c\n", optopt);
                break;

            /* should not be here */
            default:
                break;
        }
    }

    g_cuda_ctrl.dataByte = sizeof(long) * (long)(g_cuda_ctrl.stride) * (long)(g_cuda_ctrl.size) * (long)(g_cuda_ctrl.globalSize) / (long)(g_cuda_ctrl.localSize);
    g_cuda_ctrl.offset = (long)(g_cuda_ctrl.stride) * (long)(g_cuda_ctrl.size);

    fprintf(stderr, "Total buffer size: %ld\n", g_cuda_ctrl.dataByte);

    free (short_options);
}

void HostDataCreation(long* &hostArray)
{
    hostArray = (long*) malloc (g_cuda_ctrl.dataByte);

    for (int i = 0 ; i < g_cuda_ctrl.size * g_cuda_ctrl.stride * g_cuda_ctrl.globalSize / g_cuda_ctrl.localSize ; i++)
        hostArray[i] = 0;
}

int main(int argc, char* argv[])
{
    long* hostArray = NULL;
    long* devArray;
	hipEvent_t before, start, end;
 	float kernelTime;
    hipDeviceProp_t devProp;
    //PAPIWrapper papi_ctrl;


    CommandParser(argc, argv);

    for (int i = 0 ; i < 1 ; i ++)
    {
        HostDataCreation(hostArray);
        hipSetDevice(g_cuda_ctrl.device_id);
        hipGetDeviceProperties(&devProp, g_cuda_ctrl.device_id);
        //cout << "Device selected: " << devProp.name << endl;

        //papi_ctrl.AddEvent(2, strdup("cuda:::device:1:inst_executed"), strdup("cuda:::device:1:uncached_global_load_transaction"));
        hipEventCreate(&before);
        hipEventCreate(&start);
        hipEventCreate(&end);

        hipMalloc((void **)&devArray, g_cuda_ctrl.dataByte);
        hipMemcpy(devArray, hostArray, g_cuda_ctrl.dataByte, hipMemcpyHostToDevice);

        dim3 dimGrid(g_cuda_ctrl.globalSize/g_cuda_ctrl.localSize);
        dim3 dimBlock(g_cuda_ctrl.localSize);

        GeneratePattern <<<dimGrid, dimBlock>>>(devArray, g_cuda_ctrl.size, g_cuda_ctrl.stride, g_cuda_ctrl.interval);
        hipMemcpy(hostArray, devArray, g_cuda_ctrl.dataByte, hipMemcpyDeviceToHost);

        ////cout << hex << hostArray[0] << endl;

        hipEventRecord(before, 0);
        hipEventSynchronize(before);

        //papi_ctrl.Start();
        hipEventRecord(start, 0);

        Processing  <<<dimGrid, dimBlock>>>(devArray, g_cuda_ctrl.iteration, g_cuda_ctrl.offset, g_cuda_ctrl.interval);

        hipEventRecord(end, 0);
        hipEventSynchronize(end); 
        //papi_ctrl.Stop();

        hipMemcpy(hostArray, devArray, g_cuda_ctrl.dataByte, hipMemcpyDeviceToHost);

        hipEventElapsedTime(&kernelTime, start, end);
        cout << "Execution Time (s): " << kernelTime / 1000 << endl;

        if (0)
        {
            long *currArray;
            for (int i = 0 ; i < g_cuda_ctrl.globalSize/g_cuda_ctrl.localSize ; i ++)
            {
                currArray = hostArray + i * g_cuda_ctrl.stride * g_cuda_ctrl.size;
                for (int j = 0 ; j < g_cuda_ctrl.stride * g_cuda_ctrl.size ; j ++)
                {
                    cout << currArray[j] << " ";
                }
                cout << endl;
            }
        }

        free(hostArray);
        hipFree(devArray);
    }
    return 0;
}
