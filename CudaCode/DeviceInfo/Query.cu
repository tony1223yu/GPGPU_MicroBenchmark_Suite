#include <stdio.h>
 
// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
	printf("%s\n", devProp.name);
	printf("Major Compute Capability:      %d\n",  devProp.major);
    	printf("Minor Compute Capability:      %d\n",  devProp.minor);
    	printf("Total global memory:           %u bytes\n",  devProp.totalGlobalMem);
	printf("Total constant memory:         %u bytes\n",  devProp.totalConstMem);
    	printf("Total shared memory per block: %u bytes\n",  devProp.sharedMemPerBlock);
    	printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    	printf("Warp size:                     %d\n",  devProp.warpSize);
	printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
	printf("Threeads per multiprocessors:  %d\n",  devProp.maxThreadsPerMultiProcessor);
    	printf("Threads per block:             %d\n",  devProp.maxThreadsPerBlock);
    	printf("Dimension of block:            %d x %d x %d\n",  devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    	printf("Dimension of grid:             %d x %d x %d\n",  devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);
	printf("Memory pitch:                  %u bytes\n",  devProp.memPitch);
	printf("Texture alignment:             %u bytes\n",  devProp.textureAlignment);
   	printf("Clock rate:                    %d\n",  devProp.clockRate);
    	printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    	printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
}
 
int main()
{
  	// Number of CUDA devices
    	int devCount;
    	hipGetDeviceCount(&devCount);
    	printf("CUDA Device Query...\n");
    	printf("There are %d CUDA devices.\n", devCount);
 
    	// Iterate through devices
    	for (int i = 0; i < devCount; ++i)
    	{
        	// Get device properties
        	printf("\nCUDA Device #%d : ", i);
        	hipDeviceProp_t devProp;
        	hipGetDeviceProperties(&devProp, i);
        	printDevProp(devProp);
    	}
	printf("\n");
 
    	return 0;
}
